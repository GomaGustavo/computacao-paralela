/*
Sequencial
real    1m11.421s
user    1m10.983s
sys     0m0.232s

Paralelo
real    0m40.724s
user    2m33.424s
sys     0m3.183s

Paralelo - GPU - OpenMP
real    0m4.863s
user    0m3.624s
sys     0m1.211s

Paralelo - GPU - CUDA
real    0m0.442s
user    0m0.174s
sys     0m0.264s

=================================================================================

OpenMP:
    ==8143== Event result:
    Invocations                                Event Name         Min         Max         Avg       Total
    Device "GeForce GT 1030 (0)"
        Kernel: mm$_omp_fn$0
            1                            warps_launched          72          72          72          72

    ==8143== Metric result:
    Invocations                               Metric Name                        Metric Description         Min         Max         Avg
    Device "GeForce GT 1030 (0)"
        Kernel: mm$_omp_fn$0
            1                 warp_execution_efficiency                 Warp Execution Efficiency      86.81%      86.81%      86.81%

CUDA:
    ==8528== Profiling result:
    No events/metrics were profiled.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void mm(double *a, double *b, double *c, int width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    double sum = 0;
    for (int k = 0; k < width; k++) {
        double x = a[i * width + k];
        double y = b[k * width + j];
        sum += x * y;
    }

    c[i * width + j] = sum;
}

int main(){

    int width = 2000;
    int size = width * width * sizeof(double);

    double *a = (double*) malloc(size);
    double *b = (double*) malloc(size);
    double *c = (double*) malloc(size);

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            a[i * width + j] = i;
            b[i * width + j] = j;
            c[i * width + j] = 0;
        }
    }

    int block_size = 44;
    int grid_size = ((width - 1) / block_size) + 1;

    double *d_a, *d_b, *d_c;

    hipMalloc((void **) &d_a, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_b, size);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_c, size);
    
    dim3 dimGrid(grid_size, grid_size, 1);
    dim3 dimBlock(block_size, block_size, 1);

    mm<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, width);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // for(int i = 0; i < width; i++) {
    //     for(int j = 0; j < width; j++) {
    //         int index = i * width + j;
    //         printf("\n c[%d][%d] = %f", i, j, c[index]);
    //     }
    // }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}